#include "hip/hip_runtime.h"
#include "matrix.cuh"

Matrix::Matrix() {};

Matrix::Matrix(int row, int col) {
    this->height = row;
    this->width = col;
    this->elements = (double *)malloc(row * col * sizeof(double));
}

void Matrix::shuffle(vector<int> ridx) {
    Matrix temp = Matrix(this->height, this->width);
    int i, j, col = this->width;
    for (i = 0; i < this->height; i++)
        for (j = 0; j < this->width; j++)
            temp.elements[i * col + j] = this->elements[ridx[i] * col + j];
    for (i = 0; i < this->height; i++)
        for (j = 0; j < this->width; j++)
        this->elements[i * col + j] = temp.elements[i * col + j];
}

void initialize(Matrix *A, double s) {
    if (s == 0) {
        for (int i = 0; i < A->width * A->height; i++)
            A->elements[i] = 0;
    }
    else {  
        random_device rd;
        default_random_engine gen {rd()};
        normal_distribution<double> dis(0, 1);
        for (int i = 0; i < A->width * A->height; i++) {
            A->elements[i] = dis(gen) / s;
        }
    }
}

void dataCopy(Matrix *A, Matrix B, int s, int t, bool expand) {
    int i, j, width = A->width;
    if (expand == false)
        for (i = 0; i < (t - s); i++)
            for (j = 0; j < width; j++)
                A->elements[i * width + j] = B.elements[((i + s) % B.height) * width + j];
    if (expand == true)
        for (i = 0; i < (t - s); i++)
            for (j = 0; j < 10; j++)
                if (j == B.elements[(i + s) % B.height])
                    A->elements[i * 10 + j] = 1;
                else
                    A->elements[i * 10 + j] = 0;
    
}

double sigmoid(double x) {
    return 1 / (1 + exp(-x));
}

__device__ double getElement(Matrix *A, int row, int col) {
	return A->elements[row * A->width + col];
}

__device__ void setElement(Matrix *A, int row, int col, double value) {
	A->elements[row * A->width + col] = value;
}

__global__ void matDotKernel(Matrix *A, Matrix *B, Matrix *C, bool trans1, bool trans2) {
	double Cvalue = 0.0;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
    if (trans1 == false && trans2 == false)
        if (row < A->height && col < B->width) {
	        for (int i = 0; i < A->width; ++i)
		        Cvalue += getElement(A, row, i) * getElement(B, i, col);
            setElement(C, row, col, Cvalue);
        }
    if (trans1 == true)
        if (row < A->width && col < B->width) {
            for (int i = 0; i < A->height; i++)
                Cvalue += getElement(A, i, row) * getElement(B, i, col);
            setElement(C, row, col, Cvalue);
        }
    if (trans2 == true)
    if (row < A->height && col < B->height) {
        for (int i = 0; i < A->width; i++)
            Cvalue += getElement(A, row, i) * getElement(B, col, i);
        setElement(C, row, col, Cvalue);
    }      
}

__global__ void matMulKernel(Matrix *A, Matrix *B, Matrix *C) {
    double Cvalue = 0.0;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
    if (row < A->height && col < A->width) {
        Cvalue = getElement(A, row, col) * getElement(B, row, col);
        setElement(C, row, col, Cvalue);
    }  
}

__global__ void matMulKernel(Matrix *A, double k) {
    double Avalue = 0.0;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
    if (row < A->height && col < A->width) {
        Avalue = getElement(A, row, col) * k;
        setElement(A, row, col, Avalue);
    }    
}

__global__ void matPlusKernel(Matrix *A, Matrix *B, Matrix *C) {
    double Cvalue = 0.0;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
    if (B->height == 1)
        if (row < A->height && col < A->width) {
            Cvalue = getElement(A, row, col) + getElement(B, 0, col);
            setElement(C, row, col, Cvalue);
        }
}

__global__ void matSubKernel(Matrix *A, Matrix *B, Matrix *C) {
    double Cvalue = 0.0;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
    if (row < A->height && col < A->width) {
        Cvalue = getElement(A, row, col) - getElement(B, row, col);
        setElement(C, row, col, Cvalue);
    } 
}

__global__ void matSubKernel(double k, Matrix *A, Matrix *B) {
    double Bvalue = 0.0;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
    if (row < A->height && col < A->width) {
        Bvalue = k - getElement(A, row, col);
        setElement(B, row, col, Bvalue);
    }    
}

__global__ void matReLUKernel(Matrix *A) {
    double Avalue = 0, temp = 0;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
    if (row < A->height && col < A->width) {
        temp = getElement(A, row, col);
        if (temp > 0)
            Avalue = temp;
        else
            Avalue = 0;
        setElement(A, row, col, Avalue);
    }   
}

__global__ void matDerReLUKernel(Matrix *A) {
    double Avalue = 0, temp = 0;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
    if (row < A->height && col < A->width) {
        temp = getElement(A, row, col);
        if (temp > 0)
            Avalue = 1;
        else
            Avalue = 0;
        setElement(A, row, col, Avalue);
    }       
}

__global__ void matTanhKernel(Matrix *A) {
    double Avalue = 0, temp = 0;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
    if (row < A->height && col < A->width) {
        temp = 1.0 / (1 + exp((-2) * getElement(A, row, col)));
        Avalue = 2 * temp - 1;
        setElement(A, row, col, Avalue);
    }
}

__global__ void matExpKernel(Matrix *A) {
    double Avalue = 0;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
    if (row < A->height && col < A->width) {
        Avalue = exp(getElement(A, row, col));
        setElement(A, row, col, Avalue);
    }
}

__global__ void matPowKernel(Matrix *A, double k) {
    double Avalue = 0;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
    if (row < A->height && col < A->width) {
        Avalue = pow(getElement(A, row, col), k);
        setElement(A, row, col, Avalue);
    }   
}

__global__ void matSumKernel(Matrix *A, Matrix *B, int axis) {
    double Bvalue = 0;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
    if (axis == 1) {
        if (row < A->height && col < A->width) {
                for (int i = 0; i < A->width; i++)
                    Bvalue += getElement(A, row, i);
                setElement(B, row, 0, Bvalue);
        }        
    }
    if (axis == 0) {
        if (row < A->height && col < A->width) {
            for (int i = 0; i < A->height; i++)
                Bvalue += getElement(A, i, col);
            setElement(B, 0, col, Bvalue);
        }
    }
}

__global__ void matDivKernel(Matrix *A, Matrix *B) {
    double Avalue = 0;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
    if (row < A->height && col < A->width) {
        Avalue = getElement(A, row, col) / getElement(B, row, 0);
        setElement(A, row, col, Avalue);
    }  
}

__global__ void matcountEqual1(Matrix *A, Matrix *B, int *cnt) {
    int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
    if (row < A->height && col < A->width)
        if ((getElement(A, row, col) == getElement(B, row, col)) && (getElement(A, row, col) == 1))
            *cnt = *cnt + 1;
}
